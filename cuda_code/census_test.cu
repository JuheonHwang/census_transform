#include "hip/hip_runtime.h"
#include "census.cuh"
#include <stdio.h>
#define POW(X) ((X)*(X))

__global__ void VectorCensusTransform(uchar* censusTransform, uchar* image, const int census_size, const int census_filter_size, const int img_h, const int img_w)
{
	const int census_half = census_filter_size / 2;
	const int census_size_divide_by_8 = (int)ceil(census_size / 8.0f);
	const int census_filter_size_divide_by_8 = (int)ceil(census_filter_size / 8.0f);
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < census_size_divide_by_8 * img_h * img_w; idx += blockDim.x * gridDim.x) {
		int i = idx / (img_w * census_size_divide_by_8); int without_i = idx % (img_w * census_size_divide_by_8);
		int j = without_i / census_size_divide_by_8; int without_ij = without_i % census_size_divide_by_8;

		int c_i = without_ij / census_filter_size_divide_by_8;
		int without_ij_c_i = without_ij % census_filter_size_divide_by_8;

		uchar census = 0;
		uchar image_center = image[img_w * i + j];

		for (int bit_idx = 0; bit_idx < 8; bit_idx++) {
			int c_j = without_ij_c_i * 8 + bit_idx;

			if (c_j >= census_filter_size)
				break;

			int c_i_2_image_i = min(max(c_i - census_half + i, 0), img_h - 1);
			int c_j_2_image_j = min(max(c_j - census_half + j, 0), img_w - 1);

			census <<= 1;
			census |= (image_center < image[img_w * c_i_2_image_i + c_j_2_image_j]);
		}
		censusTransform[idx] = census;
	}
}

__global__ void VectorCensusXOR_N_Sum(ushort* Census_sum, uchar* leftCensus, uchar* rightCensus, const int census_size, const int census_filter_size, const int img_h, const int img_w, int disp)
{
	const int census_size_divide_by_8 = (int)ceil(census_size / 8.0f);
	const int census_filter_size_divide_by_8 = (int)ceil(census_filter_size / 8.0f);
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		ushort xor_summation = 0;

		int i = idx / img_w;
		int j = idx % img_w;

		for (int without_ij = 0; without_ij < census_size_divide_by_8; without_ij++) {
			int c_i = without_ij / census_filter_size_divide_by_8;
			int without_ij_c_i = without_ij % census_filter_size_divide_by_8;
			int diff = min(max(j + disp, 0), img_w - 1);

            uchar XOR_byte = leftCensus[i * (img_w * census_size_divide_by_8) + j * census_size_divide_by_8 + c_i * census_filter_size_divide_by_8 + without_ij_c_i]
				^ rightCensus[i * (img_w * census_size_divide_by_8) + diff * census_size_divide_by_8 + c_i * census_filter_size_divide_by_8 + without_ij_c_i];
			xor_summation += (XOR_byte & 0x01) | ((XOR_byte >> 1) & 0x01) | ((XOR_byte >> 2) & 0x01) | ((XOR_byte >> 3) & 0x01) | ((XOR_byte >> 4) & 0x01) | ((XOR_byte >> 5) & 0x01) | ((XOR_byte >> 6) & 0x01) | ((XOR_byte >> 7) & 0x01);

		}
		Census_sum[idx] = xor_summation;
	}
}


__global__ void VectorBox_N_Cost(uint* minCosts, ushort* minDispValue, uint* cost_dummy, ushort* Census_sum, const int aggreate_filter_size, const int img_h, const int img_w, int disp)
{
	const int aggreate_half = (aggreate_filter_size - 1) / 2;

	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		uint cost_summation = 0;

		int i = idx / img_w;
		int j = idx % img_w;

		const int height_begin = max(i - aggreate_half, 0);
		const int height_end = min(i + aggreate_half + 1, img_h);
		const int width_begin = max(j - aggreate_half, 0);
		const int width_end = min(j + aggreate_half + 1, img_w);

		for (int ii = height_begin; ii < height_end; ii++) {
			for (int jj = width_begin; jj < width_end; jj++) {
				cost_summation += Census_sum[ii * img_w + jj];
			}
		}

		if (disp == 0)
		{
			cost_dummy[1 * img_h * img_w + idx] = cost_summation;
			cost_dummy[2 * img_h * img_w + idx] = cost_summation;
		}
		else
		{
			cost_dummy[2 * img_h * img_w + idx] = cost_summation;

			if (cost_dummy[1 * img_h * img_w + idx] < minCosts[1 * img_h * img_w + idx])
			{
				minDispValue[idx] = (disp - 1);

				minCosts[0 * img_h * img_w + idx] = cost_dummy[0 * img_h * img_w + idx];
				minCosts[1 * img_h * img_w + idx] = cost_dummy[1 * img_h * img_w + idx];
				minCosts[2 * img_h * img_w + idx] = cost_dummy[2 * img_h * img_w + idx];
			}
		}
	}
}

__global__ void VectorShift(uint* cost_curr, uint* cost_prev, const int img_h, const int img_w)
{
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		cost_curr[0 * img_h * img_w + idx] = cost_prev[1 * img_h * img_w + idx];
		cost_curr[1 * img_h * img_w + idx] = cost_prev[2 * img_h * img_w + idx];
	}
}

__global__ void GetDispFloat(float* disp_map_float, uint* minCosts, ushort* minDispValue, const bool sub_pixel, const int img_h, const int img_w, const int max_disp)
{
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		if (sub_pixel)
		{
			float min_prev_cost = (float)(minCosts[0 * img_h * img_w + idx]);
			float min_curr_cost = (float)(minCosts[1 * img_h * img_w + idx]);
			float min_next_cost = (float)(minCosts[2 * img_h * img_w + idx]);

			disp_map_float[idx] = (float)minDispValue[idx] + (min_next_cost - min_prev_cost) / (2 * (2 * min_curr_cost - min_prev_cost - min_next_cost));
		}
		else
		{
			disp_map_float[idx] = (float)minDispValue[idx];
		}
	}
}

__global__ void GetDispInterp(float* interp_disp_map, float* target_disp_map, float* opt_disp_map, const int right2left, const int img_h, const int img_w)
{
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		int i = idx / img_w;
		int j = idx % img_w;

		float target_idx = min(max((float)j + (float)right2left * opt_disp_map[idx], 0.0f), (float)img_w - 1);
		int target_idx1 = (int)target_idx;
		int target_idx2 = min((int)target_idx + 1, img_w - 1);
		float alpha = target_idx - (int)target_idx;

		interp_disp_map[idx] = alpha * target_disp_map[i * img_w + target_idx2] + (1 - alpha) * target_disp_map[i * img_w + target_idx1];
	}
}

__global__ void ThresholdDisp(float* disp_map, float* interp_disp_map, const int threshold, const int img_h, const int img_w)
{
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		if (abs(interp_disp_map[idx] - disp_map[idx]) > threshold)
		{
			disp_map[idx] = 0;
		}
	}
}

__global__ void GetDispUchar(uchar* disp_map_uchar, float* disp_map_float, const int img_h, const int img_w, const int max_disp)
{
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < img_h * img_w; idx += blockDim.x * gridDim.x) {
		disp_map_uchar[idx] = (uchar)min((256.0f / (float)max_disp) * disp_map_float[idx], 255.0f);
	}
}

__host__ void DiffBoxImages(float* pLeftOutPfm, float* pRightOutPfm, uchar* pLeftOutImg, uchar* pRightOutImg, uchar* pimageLeft, uchar* pimageRight,
	const bool sub_pixel, const int threshold, const int census_filter_size, const int aggreate_filter_size,
	const int img_height, const int img_width, const int max_disp)
{
	uchar* imageLeft_mem;
	uchar* imageRight_mem;

	uchar* d_imageLeft_mem;
	uchar* d_imageRight_mem;


	uchar* d_leftCensus_mem;
	uchar* d_rightCensus_mem;

	ushort* d_CensusSum_mem;

	uint* d_Cost_a_mem;
	uint* d_Cost_b_mem;

	uint* d_minCosts_mem;
	ushort* d_minDispValue_mem;

	float* d_leftDispFloat_mem;
	float* d_rightDispFloat_mem;

	float* d_left2rightDisp_mem;
	float* d_right2leftDisp_mem;

	uchar* d_leftDispUchar_mem;
	uchar* d_rightDispUchar_mem;

	const int census_size = POW(census_filter_size);


	//int i = 1;
	//for (; i < census_size; i *= 8);
	int n = static_cast<int>(ceil(census_size / 8.0f));

	// Image hipMalloc
	imageLeft_mem = (uchar*)malloc(img_height * img_width * sizeof(uchar));
	memcpy(imageLeft_mem, pimageLeft, sizeof(uchar) * img_height * img_width);

	imageRight_mem = (uchar*)malloc(img_height * img_width * sizeof(uchar));
	memcpy(imageRight_mem, pimageRight, sizeof(uchar) * img_height * img_width);

	checkCudaErrors(hipMalloc(&d_imageLeft_mem, img_height * img_width * sizeof(uchar)));
	checkCudaErrors(hipMemcpy(d_imageLeft_mem, imageLeft_mem, img_height * img_width * sizeof(uchar), hipMemcpyHostToDevice));
	free(imageLeft_mem);

	checkCudaErrors(hipMalloc(&d_imageRight_mem, img_height * img_width * sizeof(uchar)));
	checkCudaErrors(hipMemcpy(d_imageRight_mem, imageRight_mem, img_height * img_width * sizeof(uchar), hipMemcpyHostToDevice));
	free(imageRight_mem);


	// Census hipMalloc
	checkCudaErrors(hipMalloc(&d_leftCensus_mem, n * img_height * img_width * sizeof(uchar)));
	checkCudaErrors(hipMemset(d_leftCensus_mem, 0, n * img_height * img_width * sizeof(uchar)));

	checkCudaErrors(hipMalloc(&d_rightCensus_mem, n * img_height * img_width * sizeof(uchar)));
	checkCudaErrors(hipMemset(d_rightCensus_mem, 0, n * img_height * img_width * sizeof(uchar)));


	VectorCensusTransform << <128, 128 >> > (d_leftCensus_mem, d_imageLeft_mem, census_size, census_filter_size, img_height, img_width);
	VectorCensusTransform << <128, 128 >> > (d_rightCensus_mem, d_imageRight_mem, census_size, census_filter_size, img_height, img_width);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());
	checkCudaErrors(hipFree(d_imageLeft_mem)); checkCudaErrors(hipFree(d_imageRight_mem));

	// CensusSum hipMalloc
	checkCudaErrors(hipMalloc(&d_CensusSum_mem, img_height * img_width * sizeof(ushort)));

	// Costs hipMalloc
	checkCudaErrors(hipMalloc(&d_Cost_a_mem, 3 * img_height * img_width * sizeof(uint)));
	checkCudaErrors(hipMalloc(&d_Cost_b_mem, 3 * img_height * img_width * sizeof(uint)));

	checkCudaErrors(hipMalloc(&d_minCosts_mem, 3 * img_height * img_width * sizeof(uint)));
	checkCudaErrors(hipMemset(d_minCosts_mem, UINT_MAX, 3 * img_height * img_width * sizeof(uint)));

	checkCudaErrors(hipMalloc(&d_minDispValue_mem, img_height * img_width * sizeof(ushort)));

	// left
	// initialize
	VectorCensusXOR_N_Sum << <128, 128 >> > (d_CensusSum_mem, d_leftCensus_mem, d_rightCensus_mem, census_size, census_filter_size, img_height, img_width, 0);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());

	VectorBox_N_Cost << <128, 128 >> > (d_minCosts_mem, d_minDispValue_mem, d_Cost_a_mem, d_CensusSum_mem, aggreate_filter_size, img_height, img_width, 0);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());

	
	for (int disp = 1; disp < (max_disp + 1); disp++) {
		// shift
		VectorShift << <128, 128 >> > (((disp % 2) ? d_Cost_b_mem : d_Cost_a_mem), ((disp % 2) ? d_Cost_a_mem : d_Cost_b_mem), img_height, img_width);
        hipDeviceSynchronize();
		checkCUDA(__LINE__, hipGetLastError());
		
		VectorCensusXOR_N_Sum << <128, 128 >> > (d_CensusSum_mem, d_leftCensus_mem, d_rightCensus_mem, census_size, census_filter_size, img_height, img_width, min(disp, max_disp - 1));
		hipDeviceSynchronize();
		checkCUDA(__LINE__, hipGetLastError());

		VectorBox_N_Cost << <128, 128 >> > (d_minCosts_mem, d_minDispValue_mem, ((disp % 2) ? d_Cost_b_mem : d_Cost_a_mem), d_CensusSum_mem, aggreate_filter_size, img_height, img_width, disp);
		hipDeviceSynchronize();
		checkCUDA(__LINE__, hipGetLastError());
	}

	// Disp hipMalloc
	checkCudaErrors(hipMalloc(&d_leftDispFloat_mem, img_height * img_width * sizeof(float)));

	GetDispFloat << <128, 128 >> > (d_leftDispFloat_mem, d_minCosts_mem, d_minDispValue_mem, sub_pixel, img_height, img_width, max_disp);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());

	// right
	// initialize
	checkCudaErrors(hipMemset(d_minCosts_mem, UINT_MAX, 3 * img_height * img_width * sizeof(uint)));

	VectorCensusXOR_N_Sum << <128, 128 >> > (d_CensusSum_mem, d_rightCensus_mem, d_leftCensus_mem, census_size, census_filter_size, img_height, img_width, 0);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());

	VectorBox_N_Cost << <128, 128 >> > (d_minCosts_mem, d_minDispValue_mem, d_Cost_a_mem, d_CensusSum_mem, aggreate_filter_size, img_height, img_width, 0);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());

	for (int disp = 1; disp < (max_disp + 1); disp++) {
		// shift
		VectorShift << <128, 128 >> > (((disp % 2) ? d_Cost_b_mem : d_Cost_a_mem), ((disp % 2) ? d_Cost_a_mem : d_Cost_b_mem), img_height, img_width);
		hipDeviceSynchronize();
		checkCUDA(__LINE__, hipGetLastError());

		VectorCensusXOR_N_Sum << <128, 128 >> > (d_CensusSum_mem, d_rightCensus_mem, d_leftCensus_mem, census_size, census_filter_size, img_height, img_width, -min(disp, max_disp - 1));
		hipDeviceSynchronize();
		checkCUDA(__LINE__, hipGetLastError());

		VectorBox_N_Cost << <128, 128 >> > (d_minCosts_mem, d_minDispValue_mem, ((disp % 2) ? d_Cost_b_mem : d_Cost_a_mem), d_CensusSum_mem, aggreate_filter_size, img_height, img_width, disp);
		hipDeviceSynchronize();
		checkCUDA(__LINE__, hipGetLastError());
	}
	checkCudaErrors(hipFree(d_leftCensus_mem)); checkCudaErrors(hipFree(d_rightCensus_mem));
	checkCudaErrors(hipFree(d_CensusSum_mem));

	// Disp hipMalloc
	checkCudaErrors(hipMalloc(&d_rightDispFloat_mem, img_height * img_width * sizeof(float)));

	GetDispFloat << <128, 128 >> > (d_rightDispFloat_mem, d_minCosts_mem, d_minDispValue_mem, sub_pixel, img_height, img_width, max_disp);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());
	checkCudaErrors(hipFree(d_minCosts_mem));
	checkCudaErrors(hipFree(d_minDispValue_mem));

	hipMalloc(&d_left2rightDisp_mem, img_height * img_width * sizeof(float));
	hipMalloc(&d_right2leftDisp_mem, img_height * img_width * sizeof(float));

	GetDispInterp << <128, 128 >> > (d_right2leftDisp_mem, d_rightDispFloat_mem, d_leftDispFloat_mem, 1, img_height, img_width);
	GetDispInterp << <128, 128 >> > (d_left2rightDisp_mem, d_leftDispFloat_mem, d_rightDispFloat_mem, -1, img_height, img_width);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());

	ThresholdDisp << <128, 128 >> > (d_leftDispFloat_mem, d_right2leftDisp_mem, threshold, img_height, img_width);
	ThresholdDisp << <128, 128 >> > (d_rightDispFloat_mem, d_left2rightDisp_mem, threshold, img_height, img_width);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());
	checkCudaErrors(hipFree(d_left2rightDisp_mem)); checkCudaErrors(hipFree(d_right2leftDisp_mem));


	checkCudaErrors(hipMalloc(&d_leftDispUchar_mem, img_height * img_width * sizeof(uchar)));
	checkCudaErrors(hipMalloc(&d_rightDispUchar_mem, img_height * img_width * sizeof(uchar)));

	GetDispUchar << <128, 128 >> > (d_leftDispUchar_mem, d_leftDispFloat_mem, img_height, img_width, max_disp);
	GetDispUchar << <128, 128 >> > (d_rightDispUchar_mem, d_rightDispFloat_mem, img_height, img_width, max_disp);
	hipDeviceSynchronize();
	checkCUDA(__LINE__, hipGetLastError());
	hipMemcpy(pLeftOutPfm, d_leftDispFloat_mem, img_height * img_width * sizeof(float), hipMemcpyDeviceToHost);
	checkCudaErrors(hipFree(d_leftDispFloat_mem));
	hipMemcpy(pRightOutPfm, d_rightDispFloat_mem, img_height * img_width * sizeof(float), hipMemcpyDeviceToHost);
	checkCudaErrors(hipFree(d_rightDispFloat_mem));


	hipMemcpy(pLeftOutImg, d_leftDispUchar_mem, img_height * img_width * sizeof(uchar), hipMemcpyDeviceToHost);
	checkCudaErrors(hipFree(d_leftDispUchar_mem));

	hipMemcpy(pRightOutImg, d_rightDispUchar_mem, img_height * img_width * sizeof(uchar), hipMemcpyDeviceToHost);
	checkCudaErrors(hipFree(d_rightDispUchar_mem));
}

__host__ void checkCUDA(const int lineNumber, hipError_t status) {
	if (status != hipSuccess) {
		fprintf(stderr, "CUDA failure at LINE %d : %s - %s\n", lineNumber, hipGetErrorName(status), hipGetErrorString(status));
		FatalError(lineNumber);
	}
}

__host__ void FatalError(const int lineNumber) {
	fprintf(stderr, "FatalError");
	if (lineNumber != 0) fprintf(stderr, " at LINE %d", lineNumber);
	fprintf(stderr, ". Program Terminated.\n");
	hipDeviceReset();
	exit(EXIT_FAILURE);
}